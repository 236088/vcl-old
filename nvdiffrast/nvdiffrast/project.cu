#include "hip/hip_runtime.h"
#include "project.h"

void Project::init(ProjectParams& pp, float* mat, Attribute& vec, int dimention) {
	pp.kernel.size = vec.vboNum;
	pp.kernel.dimention = dimention;
	pp.block = getBlock(vec.vboNum, 1);
	pp.grid = getGrid(pp.block, vec.vboNum, 1);
	pp.kernel.vec = vec.vbo;
	pp.kernel.mat = mat;
	CUDA_ERROR_CHECK(hipMalloc(&pp.kernel.out, vec.vboNum * dimention * sizeof(float)));
}

__global__ void ProjectionForwardKernel(const ProjectKernelParams pp) {
	int pidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pidx >= pp.size)return;
	float3 v = ((float3*)pp.vec)[pidx];
	for (int i = 0; i < pp.dimention; i++) {
		pp.out[pidx * pp.dimention + i] = pp.mat[i] * v.x + pp.mat[4 + i] * v.y + pp.mat[8 + i] * v.z + pp.mat[12 + i];
	}
}

void Project::forward(ProjectParams& pp) {
	void* args[] = { &pp.kernel };
	CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(ProjectionForwardKernel), pp.grid, pp.block, args, 0, NULL));
}

void Project::init(ProjectParams& pp, Attribute& vec, float* dLdout) {
	pp.grad.out = dLdout;
	CUDA_ERROR_CHECK(hipMalloc(&pp.grad.vec, vec.vboNum * 3 * sizeof(float)));
}


__global__ void ProjectionBackwardKernel(const ProjectKernelParams pp, const ProjectKernelGradParams grad) {
	int pidx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pidx >= pp.size)return;
	for (int i = 0; i < pp.dimention; i++) {
		grad.vec[pidx * 3] += pp.mat[i] * grad.out[pidx * pp.dimention + i];
		grad.vec[pidx * 3 + 1] += pp.mat[4 + i] * grad.out[pidx * pp.dimention + i];
		grad.vec[pidx * 3 + 2] += pp.mat[8 + i] * grad.out[pidx * pp.dimention + i];
	}
}

void Project::backward(ProjectParams& pp) {
	CUDA_ERROR_CHECK(hipMemset(pp.grad.vec, 0, pp.kernel.size * 3 * sizeof(float)));
	void* args[] = { &pp.kernel,&pp.grad };
	CUDA_ERROR_CHECK(hipLaunchKernel(reinterpret_cast<const void*>(ProjectionBackwardKernel), pp.grid, pp.block, args, 0, NULL));
}
