#include "hip/hip_runtime.h"
#include "material.h"

void Material::init(MaterialParams& mp, RenderingParams& p, RasterizeParams& rp, InterpolateParams& pos, InterpolateParams& normal, float* in) {
    mp.kernel.width = p.width;
    mp.kernel.height = p.height;
    mp.kernel.depth = p.depth;
    mp.kernel.pos = pos.kernel.out;
    mp.kernel.normal = normal.kernel.out;
    mp.kernel.rast = rp.kernel.out;
    mp.kernel.in = in;
    hipMalloc(&mp.kernel.out, p.width * p.height * 3 * sizeof(float));
    mp.block = getBlock(p.width, p.height);
    mp.grid = getGrid(mp.block, p.width, p.height);
}

void Material::init(MaterialParams& mp, float3* eye, int lightNum, float3* lightpos, float3* lightintensity, float3 ambient, float Ka, float Kd, float Ks, float shininess) {
    mp.kernel.eye = eye;
    mp.kernel.lightNum = lightNum;
    hipMalloc(&mp.kernel.lightpos, lightNum * sizeof(float3));
    hipMemcpy(mp.kernel.lightpos, lightpos, lightNum * sizeof(float3), hipMemcpyHostToDevice);
    hipMalloc(&mp.kernel.lightintensity, (lightNum + 1) * sizeof(float3));
    hipMemcpy(mp.kernel.lightintensity, &ambient, sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(&mp.kernel.lightintensity[1], lightintensity, lightNum * sizeof(float3), hipMemcpyHostToDevice);
    hipMalloc(&mp.kernel.params, 4 * sizeof(float));
    float params[4]{ Ka, Kd,  Ks,  shininess };
    hipMemcpy(mp.kernel.params, params, 4 * sizeof(float), hipMemcpyHostToDevice);
}

__global__ void MaterialForwardKernel(const MaterialKernelParams mp) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= mp.width || py >= mp.height || pz >= mp.depth)return;
    int pidx = px + mp.width * (py + mp.height * pz);

    if (mp.rast[pidx * 4 + 3] < 1.f) return;

    float3 pos = ((float3*)mp.pos)[pidx];
    float3 n = ((float3*)mp.normal)[pidx];
    float3 v = *(float3*)&mp.eye - pos;
    v *= (1.f / sqrt(dot(v, v)));
    float3 diffuse = make_float3(0.f, 0.f, 0.f);
    float3 specular = make_float3(0.f, 0.f, 0.f);
    for (int i = 0; i < mp.lightNum; i++) {
        float3 lightpos = mp.lightpos[i];
        float3 l = lightpos - pos;
        l *= (1.f / sqrt(dot(l, l)));
        float ln = dot(l, n);
        float3 r = 2.f * ln * n - l;
        float rv = dot(r, v);
        float3 intensity = mp.lightintensity[i + 1];
        diffuse += intensity * max(ln, 0.f);
        float powrv = pow(max(rv, 0.f), mp.params[3]);
        AddNaNcheck(specular.x, intensity.x * powrv);
        AddNaNcheck(specular.y, intensity.y * powrv);
        AddNaNcheck(specular.z, intensity.z * powrv);
    }
    float Ka = mp.params[0];
    float Kd = mp.params[1];
    float Ks = mp.params[2];
    ((float3*)mp.out)[pidx] = ((float3*)mp.in)[pidx] * (Ka * mp.lightintensity[0] + Kd * diffuse + Ks * specular);
}

void Material::forward(MaterialParams& mp) {
    hipMemset(mp.kernel.out, 0, mp.kernel.width * mp.kernel.height * 3 * sizeof(float));
    void* args[] = { &mp.kernel};
    hipLaunchKernel(reinterpret_cast<const void*>(MaterialForwardKernel), mp.grid, mp.block, args, 0, NULL);
}

void Material::init(MaterialParams& mp, RenderingParams& p, float* dLdout) {
    mp.grad.out = dLdout;
    hipMalloc(&mp.grad.in, p.width * p.height * 3 * sizeof(float));
    hipMalloc(&mp.grad.lightpos, mp.kernel.lightNum * sizeof(float3));
    hipMalloc(&mp.grad.lightintensity, (mp.kernel.lightNum + 1) * sizeof(float3));
    hipMalloc(&mp.grad.params, 4 * sizeof(float));
}

__global__ void MaterialBackwardKernel(const MaterialKernelParams mp, const MaterialGradParams grad) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= mp.width || py >= mp.height || pz >= mp.depth)return;
    int pidx = px + mp.width * (py + mp.height * pz);

    if (mp.rast[pidx * 4 + 3] < 1.f) return;

    float3 pos = ((float3*)mp.pos)[pidx];
    float3 n = ((float3*)mp.normal)[pidx];
    float3 v = *(float3*)&mp.eye - pos;
    v *= (1. / sqrt(dot(v, v)));
    float3 diffuse = make_float3(0.f, 0.f, 0.f);
    float3 specular = make_float3(0.f, 0.f, 0.f);
    float dshine = 0.f;
    for (int i = 0; i < mp.lightNum; i++) {
        float3 light = mp.lightpos[i];
        //dl/dlight=1
        //dln/dl_=-n_*l_/dot(l,l)/sqrt(dot(l, l))
        float3 l = light - pos;
        l *= (1.f / sqrt(dot(l, l)));
        float ln = dot(l, n);
        //dr/dl_=2*n_*n-1
        //drv/dr=v
        //dspec/drv=shininess*pow(rv,shininess-1)
        float3 r = 2.f * ln * n - l;
        float rv = dot(r, v);
        float3 intensity = mp.lightintensity[i + 1];
        diffuse += intensity * max(ln, 0.f);
        float powrv = pow(max(rv, 0.f), mp.params[3]);
        AddNaNcheck(specular.x, intensity.x * powrv);
        AddNaNcheck(specular.y, intensity.y * powrv);
        AddNaNcheck(specular.z, intensity.z * powrv);
        AddNaNcheck(dshine, (intensity.x + intensity.y + intensity.z) * log(max(rv, 0.f)) * powrv);
    }
    float Ka = mp.params[0];
    float Kd = mp.params[1];
    float Ks = mp.params[2];
    float3 dLdout = ((float3*)grad.out)[pidx];
    float3 din = dLdout * ((float3*)mp.in)[pidx];
    atomicAdd(&grad.params[0], dot(mp.lightintensity[0], din));
    atomicAdd(&grad.params[1], dot(din,diffuse));
    atomicAdd(&grad.params[2], dot(din,specular));
    atomicAdd(&grad.params[3], Ks * dshine);
    ((float3*)grad.in)[pidx] = dLdout * (Ka * mp.lightintensity[0] + Kd * diffuse + Ks * specular);
}

void Material::backward(MaterialParams& mp) {
    hipMemset(mp.grad.in, 0, mp.kernel.width * mp.kernel.height * 3 * sizeof(float));
    hipMemset(&mp.grad.lightpos, 0, mp.kernel.lightNum * sizeof(float3));
    hipMemset(&mp.grad.lightintensity, 0, (mp.kernel.lightNum + 1) * sizeof(float3));
    hipMemset(&mp.grad.params, 0, 4 * sizeof(float));
    void* args[] = { &mp.kernel, &mp.grad };
    hipLaunchKernel(reinterpret_cast<const void*>(MaterialBackwardKernel), mp.grid, mp.block, args, 0, NULL);
}