#include "hip/hip_runtime.h"
#include "filter.h"

void Filter::init(FilterParams& fp, RenderingParams& p, float* in, int channel, int count) {
	fp.kernel.width = p.width;
	fp.kernel.height = p.height;
	fp.kernel.depth = p.depth;
	fp.kernel.channel = channel;
	fp.kernel.count = count;
	fp.kernel.in = in;
	float filter[9] = { 0.0625f,0.125f,0.0625f,0.125f,0.25f,0.125f,0.0625f,0.125f,0.0625f };
	hipMalloc(&fp.kernel.filter, 9 * sizeof(float));
	hipMemcpy(fp.kernel.filter, filter, 9 * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc(&fp.kernel.out, p.width * p.height * channel * sizeof(float));
	hipMalloc(&fp.kernel.buf, p.width * p.height * channel * sizeof(float));
	fp.block = getBlock(p.width, p.height);
	fp.grid = getGrid(fp.block, p.width, p.height);
}

__global__ void FilterForwardKernel(const FilterKernelParams fp) {
	int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;
	int pz = blockIdx.z;
	if (px >= fp.width || py >= fp.height || pz >= fp.depth)return;
	int pidx = px + fp.width * (py + fp.height * pz);

	int si = 0 < px ? -1 : 0;
	int ei = px < fp.width - 1 ? 1 : 0;
	int sj = 0 < py ? -1 : 0;
	int ej = py < fp.height - 1 ? 1 : 0;
	for (int i = si; i <= ei; i++) {
		for (int j = sj; j <= ej; j++) {
			for (int k = 0; k < fp.channel; k++) {
				int idx = (px + i) + fp.width * (py + j);
				fp.out[pidx * fp.channel + k] += fp.buf[idx * fp.channel + k] * fp.filter[i + 3 * j + 4];
			}
		}
	}
}

void Filter::forward(FilterParams& fp) {
	hipMemcpy(fp.kernel.buf, fp.kernel.in, fp.kernel.width * fp.kernel.height * fp.kernel.channel * sizeof(float), hipMemcpyDeviceToDevice);
	void* args[] = { &fp.kernel };
	for (int i = 0; i < fp.kernel.count; i++) {
		hipMemset(fp.kernel.out, 0, fp.kernel.width * fp.kernel.height * fp.kernel.channel * sizeof(float));
		hipLaunchKernel(reinterpret_cast<const void*>(FilterForwardKernel), fp.grid, fp.block, args, 0, NULL);
		hipMemcpy(fp.kernel.buf, fp.kernel.out, fp.kernel.width * fp.kernel.height * fp.kernel.channel * sizeof(float), hipMemcpyDeviceToDevice);
	}
}

void Filter::init(FilterParams& fp, RenderingParams& p, float* dLdout) {
	fp.grad.out= dLdout;
	hipMalloc(&fp.grad.in, fp.kernel.width * fp.kernel.height * fp.kernel.channel * sizeof(float));
}

__global__ void FilterBackwardKernel(const FilterKernelParams fp,const  FilterGradParams grad) {
	int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;
	int pz = blockIdx.z;
	if (px >= fp.width || py >= fp.height || pz >= fp.depth)return;
	int pidx = px + fp.width * (py + fp.height * pz);

	int si = 0 < px ? -1 : 0;
	int ei = px < fp.width - 1 ? 1 : 0;
	int sj = 0 < py ? -1 : 0;
	int ej = py < fp.height - 1 ? 1 : 0;
	for (int i = si; i <= ei; i++) {
		for (int j = sj; j <= ej; j++) {
			for (int k = 0; k < fp.channel; k++) {
				int idx = (px + i) + fp.width * (py + j);
				grad.in[pidx * fp.channel + k] += fp.buf[idx * fp.channel + k] * fp.filter[i + 3 * j + 4];
			}
		}
	}
}

void Filter::backward(FilterParams& fp) {
	hipMemcpy(fp.kernel.buf, fp.grad.out, fp.kernel.width * fp.kernel.height * fp.kernel.channel * sizeof(float), hipMemcpyDeviceToDevice);
	void* args[] = { &fp.kernel, &fp.grad };
	for (int i = 0; i < fp.kernel.count; i++) {
		hipMemset(fp.grad.in, 0, fp.kernel.width * fp.kernel.height * fp.kernel.channel * sizeof(float));
		hipLaunchKernel(reinterpret_cast<const void*>(FilterBackwardKernel), fp.grid, fp.block, args, 0, NULL);
		hipMemcpy(fp.kernel.buf, fp.grad.in, fp.kernel.width * fp.kernel.height * fp.kernel.channel * sizeof(float), hipMemcpyDeviceToDevice);
	}
}
