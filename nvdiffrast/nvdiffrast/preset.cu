#include "preset.h"

void GLbuffer::init(GLbuffer& rb, float* buffer, int width, int height, int channel, int attachmentNum) {
	rb.width = width;
	rb.height = height;
	rb.channel = channel;
	rb.buffer = buffer;
	CUDA_ERROR_CHECK(hipHostMalloc(&rb.gl_buffer, width * height * channel * sizeof(float)));
	glGenTextures(1, &rb.id);
	glBindTexture(GL_TEXTURE_2D, rb.id);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glFramebufferTexture(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0 + attachmentNum, rb.id, 0);
}

void GLbuffer::draw(GLbuffer& rb, GLint internalformat, GLenum format, float minX, float minY, float maxX, float maxY) {
	CUDA_ERROR_CHECK(hipMemcpy(rb.gl_buffer, rb.buffer, rb.width * rb.height * rb.channel * sizeof(float), hipMemcpyDeviceToHost));
	glBindTexture(GL_TEXTURE_2D, rb.id);
	glTexImage2D(GL_TEXTURE_2D, 0, internalformat, rb.width, rb.height, 0, format, GL_FLOAT, rb.gl_buffer);
	glBegin(GL_POLYGON);
	glTexCoord2f(0.f, 0.f); glVertex2f(minX, minY);
	glTexCoord2f(0.f, 1.f); glVertex2f(minX, maxY);
	glTexCoord2f(1.f, 1.f); glVertex2f(maxX, maxY);
	glTexCoord2f(1.f, 0.f); glVertex2f(maxX, minY);
	glEnd();
}