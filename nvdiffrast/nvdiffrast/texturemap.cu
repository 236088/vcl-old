#include "hip/hip_runtime.h"
#include "common.h"
#include "texturemap.h"

__global__ void copy(unsigned char* data, const TexturemapParams tp) {
	int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;
	int pz = blockIdx.z;
	if (px >= tp.width || py >= tp.height)return;
	int pidx = px + tp.width * (py + tp.height * pz);

	for (int i = 0; i < tp.channel; i++) {
		tp.miptex[0][pidx * tp.channel + i] = (float)data[pidx * tp.channel + i] / 255.0;
	}
}

void Texturemap::loadBMP(TexturemapParams& tp, const char* path) {
	unsigned char header[54];

	FILE* file = fopen(path, "rb");
	if (!file) {
		printf("Image could not be opened\n");
		return;
	}
	if (fread(header, 1, 54, file) != 54) {
		printf("Not a correct BMP file\n");
		return;
	}
	if (header[0] != 'B' || header[1] != 'M') {
		printf("Not a correct BMP file\n");
		return;
	}
	if (*(int*)&(header[0x12]) != tp.width || *(int*)&(header[0x16]) != tp.height) {
		printf("Not match texWidth or texHeight value\n");
		return;
	}
	unsigned int dataPos = *(int*)&(header[0x0A]);
	unsigned int imageSize = *(int*)&(header[0x22]);

	if (imageSize == 0)    imageSize = tp.width * tp.height * tp.channel;
	if (dataPos == 0)      dataPos = 54;
	fseek(file, dataPos, SEEK_SET);

	unsigned char* data = new unsigned char[imageSize];
	fread(data, 1, imageSize, file);
	fclose(file);

	unsigned char* dev_data;

	hipMalloc(&dev_data, tp.width * tp.height * tp.channel * sizeof(unsigned char));
	hipMemcpy(dev_data, data, tp.width * tp.height * tp.channel * sizeof(unsigned char), hipMemcpyHostToDevice);

	copy << < tp.grid, tp.block >> > (dev_data, tp);
	hipFree(dev_data);
}

__global__ void downSampling(const TexturemapParams tp, int index, int width, int height) {
	int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;
	int pz = blockIdx.z;
	if (px >= width || py >= height)return;
	int pidx = px + width * (py + height * pz);
	px <<= 1; py <<= 1;
	width <<= 1; height <<= 1;
	int p00idx = px + width * (py + height * pz);
	int p01idx = (px + 1) + width * (py + height * pz);
	int p10idx = px + width * ((py + 1) + height * pz );
	int p11idx = (px + 1) + width * ((py + 1) + height * pz);

	for (int i = 0; i < tp.channel; i++) {
		float p00 = tp.miptex[index - 1][p00idx * tp.channel + i];
		float p01 = tp.miptex[index - 1][p01idx * tp.channel + i];
		float p10 = tp.miptex[index - 1][p10idx * tp.channel + i];
		float p11 = tp.miptex[index - 1][p11idx * tp.channel + i];

		float p = (p00 + p01 + p10 + p11) * 0.25;
		tp.miptex[index][pidx * tp.channel + i] = p;
	}
}

void Texturemap::buildMipTexture(TexturemapParams& tp) {
	int w = tp.width, h = tp.height;
	for (int i = 1; i < tp.miplevel; i++) {
		w >>= 1; h >>= 1;
		dim3 block = getBlock(w, h);
		dim3 grid = getGrid(block, w, h);
		downSampling << <grid, block >> > (tp, i, w, h);
	}
}

void Texturemap::forwardInit(TexturemapParams& tp, RenderingParams& p, RasterizeParams& rp, InterpolateParams& ip, int width, int height, int channel, int miplevel) {
	tp.miplevel = miplevel < TEX_MAX_MIP_LEVEL ? miplevel : TEX_MAX_MIP_LEVEL;
	if(((width >> tp.miplevel) << tp.miplevel) != width || ((height >> tp.miplevel) << tp.miplevel) != height){
		printf("Invalid miplevel value");
		exit(1);
	}
	tp.width = width;
	tp.height = height;
	tp.channel = channel;
	tp.rast = rp.out;
	tp.uv = ip.out;
	tp.uvDA = ip.outDA;
	tp.block = getBlock(tp.width, tp.height);
	tp.grid = getGrid(tp.block, tp.width, tp.height);

	int w = width, h = height;
	for (int i = 0; i < miplevel; i++) {
		hipMalloc(&tp.miptex[i], w * h * channel * sizeof(float));
		w >>= 1; h >>= 1;
	}
	hipMalloc(&tp.out, p.width * p.height * channel * sizeof(float));
}

__device__ __forceinline__ int4 indexFetch(const TexturemapParams tp, int level, float2 uv, float2& t) {
	int2 size = make_int2(tp.width >> level, tp.height >> level);
	t.x = uv.x * (float)size.x;
	t.y = uv.y * (float)size.y;
	int u0 = t.x<0 ? 0 : t.x>size.x - 1 ? size.x - 1 : (int)t.x;
	int u1 = t.x<1 ? 0 : t.x>size.x - 2 ? size.x - 1 : (int)t.x + 1;
	int v0 = t.y<0 ? 0 : t.y>size.y - 1 ? size.y - 1 : (int)t.y;
	int v1 = t.y<1 ? 0 : t.y>size.y - 2 ? size.y - 1 : (int)t.y + 1;
	int4 idx;
	idx.x = v0 * size.x + u0;
	idx.y = v0 * size.x + u1;
	idx.z = v1 * size.x + u0;
	idx.w = v1 * size.x + u1;
	t.x = t.x<0 ? 0 : size.x<t.x ? 1 : t.x - floor(t.x);
	t.y = t.y<0 ? 0 : size.y<t.y ? 1 : t.y - floor(t.y);
	return idx;
}

__global__ void texturemapFowardKernel(const TexturemapParams tp, const RenderingParams p) {
	int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;
	int pz = blockIdx.z;
	if (px >= p.width || py >= p.height || pz >= p.depth)return;
	int pidx = px + p.width * (py + p.height * pz);

	if (tp.rast[pidx * 4 + 3] < 1.0) return;

	float4 uvDA = ((float4*)tp.uvDA)[pidx];
	float dsdx = uvDA.x * tp.width;
	float dsdy = uvDA.y * tp.width;
	float dtdx = uvDA.z * tp.height;
	float dtdy = uvDA.w * tp.height;

	// calculate footprint
	// b is sum of 2 square sides 
	// b = (dsdx^2+dsdy^2) + (dtdx^2+dtdy^2)
	// c is square area
	// c = (dsdx * dtdy - dtdx * dsdy)^2
	// solve x^2 - bx + c = 0

	float s2 = dsdx * dsdx + dsdy * dsdy;
	float t2 = dtdx * dtdx + dtdy * dtdy;
	float a = dsdx * dtdy - dtdx * dsdy;

	float b = 0.5 * (s2 + t2);
	float c = sqrt(b * b - a * a);

	float level = 0.5 * log2f(b + c);
	int level0 = level <= 0 ? 0 : tp.miplevel - 2 <= level ? tp.miplevel - 2 : (int)floor(level);
	int level1 = level <= 1 ? 1 : tp.miplevel - 1 <= level ? tp.miplevel - 1 : (int)floor(level) + 1;
	float flevel = level <= 0 ? 0 : tp.miplevel - 1 <= level ? 1 : level - floor(level);


	float2 uv = ((float2*)tp.uv)[pidx];
	float2 uv0, uv1;
	int4 idx0 = indexFetch(tp, level0, uv, uv0);
	int4 idx1 = indexFetch(tp, level1, uv, uv1);
	for (int i = 0; i < tp.channel; i++) {
		float out = bilerp(
			tp.miptex[level0][idx0.x * tp.channel + i], tp.miptex[level0][idx0.y * tp.channel + i],
			tp.miptex[level0][idx0.z * tp.channel + i], tp.miptex[level0][idx0.w * tp.channel + i], uv0);
		if (flevel > 0) {
			float out1 = bilerp(
				tp.miptex[level1][idx1.x * tp.channel + i], tp.miptex[level1][idx1.y * tp.channel + i],
				tp.miptex[level1][idx1.z * tp.channel + i], tp.miptex[level1][idx1.w * tp.channel + i], uv1);
			out = lerp(out, out1, flevel);
		}
		tp.out[pidx * tp.channel + i] = out;
	}
}

void Texturemap::forward(TexturemapParams& tp, RenderingParams& p) {
	hipMemset(tp.out, 0, p.width * p.height * tp.channel * sizeof(float));
	texturemapFowardKernel << <p.grid, p.block >> > (tp, p);
}


__device__ __forceinline__ void calculateLevel(const TexturemapParams tp, int pidx, int& level0, int& level1, float& flevel, float4& dleveldda) {
	float4 uvDA = ((float4*)tp.uvDA)[pidx];
	float dsdx = uvDA.x * tp.width;
	float dsdy = uvDA.y * tp.width;
	float dtdx = uvDA.z * tp.height;
	float dtdy = uvDA.w * tp.height;

	float s2 = dsdx * dsdx + dsdy * dsdy;
	float t2 = dtdx * dtdx + dtdy * dtdy;
	float a = dsdx * dtdy - dtdx * dsdy;

	float b = 0.5 * (s2 + t2);
	float c2 = b * b - a * a;
	float c = sqrt(c2);


	float level = 0.5 * log2f(b + c);
	level0 = level <= 0 ? 0 : tp.miplevel - 2 <= level ? tp.miplevel - 2 : (int)floor(level);
	level1 = level <= 1 ? 1 : tp.miplevel - 1 <= level ? tp.miplevel - 1 : (int)floor(level) + 1;
	flevel = level <= 0 ? 0 : tp.miplevel - 1 <= level ? 1 : level - floor(level);

	float d = b * c + c2; // b^2 - a^2 == 0 or not if 0 then level=ln(b)
	if (abs(d) > 1e-6) {
		d = 0.72134752f / d;
		float bc = b + c;
		dleveldda = make_float4(d * (bc * dsdx - a * dtdy), d * (bc * dsdy + a * dtdx), (bc * dtdx + a * dsdy), (bc * dtdy - a * dsdx));
	}
	else {
		// if abs(b) == 0 then dsdx, dsdy, dtdx, dtdy are 0
		if (abs(b) > 1e-6) {
			d = 1 / b;
			dleveldda = make_float4(d * dsdx, d * dsdy, d * dtdx, d * dtdy);
		}
		else {
			dleveldda = make_float4(0.0, 0.0, 0.0, 0.0);
		}	
	}
}

// s_ = frac(s*width_) => d/ds = d/ds_ * width_
// t_ = frac(t*height_) => d/dt = d/dt_ * height_
// l = frac(level) => dl/dlevel = 1
//
// dL/dX = dL/dc * dc/dX
//
// dc/ds = lerp(lerp(c001-c000, c011-c010, t0) * width0, lerp(c101-c100, c111-c110, t1) * width1, l)
// dc/dt = lerp(lerp(c010-c000, c011-c001, s0) * height0, lerp(c110-c100, c111-c101, s1) * height1, l)
// dc/dlevel = -bilerp(c000,c001,c010,c011,s0,t0) + bilerp(c100,c101,c110,c111,s1,t1)
//
// dc/dc000 = (1-l) * (1-s0) * (1-t0)
// :
// :
// dc/dc111 = l * s1 * t1
// 
// 
//
// dL/dX = dL/dc * dc/dlevel * dlevel/dX
// 
// b = ((ds/dx^2+ds/dy^2) + (dt/dx^2+dt/dy^2))/2
// a = ds/dx * dt/dy - dt/dx * ds/dy
// level = ln(b + sqrt(b^2 - a^2))/2ln2
//
// dlevel/dX = 1/2ln2 * (b'+(b*b'-a*a')/sqrt(b^2-a^2))/(b+sqrt(b^2-a^2))
//           = 1/2ln2/(b * sqrt(b^2-a^2) + (b^2-a^2)) * ((sqrt(b^2-a^2) + b) * b'- a * a')
// dlevel/d(ds/dx) = 1/2ln2/(b * sqrt(b^2-a^2) + (b^2-a^2)) * ((sqrt(b^2-a^2) + b) * ds/dx - a * dt/dy)
// dlevel/d(ds/dy) = 1/2ln2/(b * sqrt(b^2-a^2) + (b^2-a^2)) * ((sqrt(b^2-a^2) + b) * ds/dy + a * dt/dx)
// dlevel/d(dt/dx) = 1/2ln2/(b * sqrt(b^2-a^2) + (b^2-a^2)) * ((sqrt(b^2-a^2) + b) * dt/dx + a * ds/dy)
// dlevel/d(dt/dy) = 1/2ln2/(b * sqrt(b^2-a^2) + (b^2-a^2)) * ((sqrt(b^2-a^2) + b) * dt/dy - a * ds/dx)
//
__global__ void texturemapBackwardKernel(const TexturemapParams tp, const RenderingParams p) {
	int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;
	int pz = blockIdx.z;
	if (px >= p.width || py >= p.height || pz >= p.depth)return;
	int pidx = px + p.width * (py + p.height * pz);
	if (tp.rast[pidx * 4 + 3] < 1.0) {
		((float2*)tp.gradUV)[pidx] = make_float2(0.0, 0.0);
		((float4*)tp.gradUVDA)[pidx] = make_float4(0.0, 0.0, 0.0, 0.0);
		return;
	}
	int level0 = 0, level1 = 0;
	float flevel = 0.0;
	float4 dleveldda;
	calculateLevel(tp, pidx, level0, level1, flevel, dleveldda);
	float2 uv = ((float2*)tp.uv)[pidx], uv0, uv1;
	float gu = 0.0, gv = 0.0, gl = 0.0;
	int4 idx0 = indexFetch(tp, level0, uv, uv0);
	int4 idx1 = indexFetch(tp, level1, uv, uv1);

	for (int i = 0; i < tp.channel; i++) {
		float dLdout = tp.dLdout[pidx * tp.channel + i];
		float check = atomicAdd(&tp.gradMipTex[level0][idx0.x * tp.channel + i], (1.0 - flevel) * (1.0 - uv0.x) * (1.0 - uv0.y) * dLdout);
		atomicAdd(&tp.gradMipTex[level0][idx0.y * tp.channel + i], (1.0 - flevel) * uv0.x * (1.0 - uv0.y) * dLdout);
		atomicAdd(&tp.gradMipTex[level0][idx0.z * tp.channel + i], (1.0 - flevel) * (1.0 - uv0.x) * uv0.y * dLdout);
		atomicAdd(&tp.gradMipTex[level0][idx0.w * tp.channel + i], (1.0 - flevel) * uv0.x * uv0.y * dLdout);
		float t00 = tp.miptex[level0][idx0.x * tp.channel + i];
		float t01 = tp.miptex[level0][idx0.y * tp.channel + i];
		float t10 = tp.miptex[level0][idx0.z * tp.channel + i];
		float t11 = tp.miptex[level0][idx0.w * tp.channel + i];

		float u = lerp(t01 - t00, t11 - t10, uv0.y) * (tp.width >> level0);
		float v = lerp(t10 - t00, t11 - t01, uv0.x) * (tp.height >> level0);
		if (flevel > 0) {
			float l = bilerp(t00, t01, t10, t11, uv0);
			atomicAdd(&tp.gradMipTex[level1][idx1.x * tp.channel + i], flevel * (1.0 - uv1.x) * (1.0 - uv1.y) * dLdout);
			atomicAdd(&tp.gradMipTex[level1][idx1.y * tp.channel + i], flevel * uv1.x * (1.0 - uv1.y) * dLdout);
			atomicAdd(&tp.gradMipTex[level1][idx1.z * tp.channel + i], flevel * (1.0 - uv1.x) * uv1.y * dLdout);
			atomicAdd(&tp.gradMipTex[level1][idx1.w * tp.channel + i], flevel * uv1.x * uv1.y * dLdout);
			t00 = tp.miptex[level1][idx1.x * tp.channel + i];
			t01 = tp.miptex[level1][idx1.y * tp.channel + i];
			t10 = tp.miptex[level1][idx1.z * tp.channel + i];
			t11 = tp.miptex[level1][idx1.w * tp.channel + i];
			u = lerp(u, lerp(t01 - t00, t11 - t10, uv1.y) * (tp.width >> level1), flevel);
			v = lerp(v, lerp(t10 - t00, t11 - t01, uv1.x) * (tp.height >> level1), flevel);
			gl += (bilerp(t00, t01, t10, t11, uv1) - l) * dLdout;
		}
		gu += u * dLdout;
		gv += v * dLdout;
	}

	((float2*)tp.gradUV)[pidx] = make_float2(gu, gv);
	((float4*)tp.gradUVDA)[pidx] = gl * dleveldda;
}

__global__ void texturemapGradSum(const TexturemapParams tp) {
	int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;
	int pz = blockIdx.z;
	if (px >= tp.width || py >= tp.height)return;
	int pidx = px + tp.width * (py + tp.height * pz);

	int w = tp.width, h = tp.height;
	for (int l = 0; l < tp.miplevel; l++) {
		int p = px + w * (py + h * pz);
		for (int i = 0; i < tp.channel; i++) {
			float g = tp.gradMipTex[l][p * tp.channel + i];
			if (!isnan(g))tp.gradTex[pidx * tp.channel + i] += g;
		}
		px >>= 1; py >>= 1;
		w >>= 1; h >>= 1;
	}
}

void Texturemap::backward(TexturemapParams& tp, RenderingParams& p) {
	hipMemset(tp.gradTex, 0, tp.width * tp.height * tp.channel * sizeof(float));
	int w = tp.width, h = tp.height;
	for (int i = 0; i < tp.miplevel; i++) {
		hipMemset(tp.gradMipTex[i], 0, w * h * tp.channel * sizeof(float));
		w >>= 1; h >>= 1;
	}
	texturemapBackwardKernel << <p.grid, p.block >> > (tp, p);
	texturemapGradSum << <tp.grid, tp.block >> > (tp);
}

void Texturemap::backwardInit(TexturemapParams& tp, RenderingParams& p, float* dLdout) {
	tp.dLdout = dLdout;
	hipMalloc(&tp.gradUV, p.height * p.height * 2 * sizeof(float));
	hipMalloc(&tp.gradUVDA, p.height * p.height * 4 * sizeof(float));
	hipMalloc(&tp.gradTex, tp.width * tp.height * tp.channel * sizeof(float));

	int w = tp.width, h = tp.height;
	for (int i = 0; i < tp.miplevel; i++) {
		hipMalloc(&tp.gradMipTex[i], w * h * tp.channel * sizeof(float));
		w >>= 1; h >>= 1;
	}
}