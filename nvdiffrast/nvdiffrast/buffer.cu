#include "hip/hip_runtime.h"
#include "buffer.h"

void Attribute::init(Attribute& attr, float* h_vbo, unsigned int* h_vao, int vboNum, int vaoNum, int dimention) {
    attr.dimention = dimention;
    attr.vboNum = vboNum;
    attr.vaoNum = vaoNum;
    hipHostMalloc(&attr.h_vbo, vboNum * dimention * sizeof(float));
    hipHostMalloc(&attr.h_vao, vaoNum * 3 * sizeof(unsigned int));
    hipMalloc(&attr.vbo, vboNum * dimention * sizeof(float));
    hipMalloc(&attr.vao, vaoNum * 3 * sizeof(float));
    hipMemcpy(attr.h_vbo, h_vbo, vboNum * dimention * sizeof(float), hipMemcpyHostToHost);
    hipMemcpy(attr.h_vao, h_vao, vaoNum * 3 * sizeof(float), hipMemcpyHostToHost);
    hipMemcpy(attr.vbo, h_vbo, vboNum * dimention * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(attr.vao, h_vao, vaoNum * 3 * sizeof(float), hipMemcpyHostToDevice);
}

void Attribute::loadOBJ(const char* path, Attribute& pos, Attribute& texel, Attribute& normal) {
    FILE* file = fopen(path, "r");
    if (file == NULL) {
        printf("Impossible to open the file !\n");
        return;
    }

    std::vector<float> tempPos, tempTexel, tempNorm;
    std::vector<unsigned int> tempPosIndex, tempTexelIndex, tempNormIndex;
    int posNum = 0, texelNum = 0, normNum = 0, indexNum = 0;
    while (1) {
        char lineHeader[128];
        int res = fscanf(file, "%s", lineHeader);
        if (res == EOF)
            break;
        if (strcmp(lineHeader, "v") == 0) {
            float v[3];
            fscanf(file, "%f %f %f\n", &v[0], &v[1], &v[2]);
            tempPos.push_back(v[0]);
            tempPos.push_back(v[1]);
            tempPos.push_back(v[2]);
            posNum++;
        }
        else if (strcmp(lineHeader, "vt") == 0) {
            float v[2];
            fscanf(file, "%f %f\n", &v[0], &v[1]);
            tempTexel.push_back(v[0]);
            tempTexel.push_back(v[1]);
            texelNum++;
        }
        else if (strcmp(lineHeader, "vn") == 0) {
            float v[3];
            fscanf(file, "%f %f %f\n", &v[0], &v[1], &v[2]);
            tempNorm.push_back(v[0]);
            tempNorm.push_back(v[1]);
            tempNorm.push_back(v[2]);
            normNum++;
        }
        else if (strcmp(lineHeader, "f") == 0 && posNum > 0) {
            unsigned int idx[9];
            if (texelNum > 0 && normNum > 0) {
                int matches = fscanf(file, "%d/%d/%d %d/%d/%d %d/%d/%d\n", &idx[0], &idx[3], &idx[6], &idx[1], &idx[4], &idx[7], &idx[2], &idx[5], &idx[8]);
                if (matches != 9) {
                    printf("File can't be read by our simple parser : ( Try exporting with other options\n");
                    return;
                }
                tempTexelIndex.push_back(idx[3] - 1);
                tempTexelIndex.push_back(idx[4] - 1);
                tempTexelIndex.push_back(idx[5] - 1);
                tempNormIndex.push_back(idx[6] - 1);
                tempNormIndex.push_back(idx[7] - 1);
                tempNormIndex.push_back(idx[8] - 1);
            }
            else if (texelNum > 0) {
                int matches = fscanf(file, "%d/%d %d/%d %d/%d\n", &idx[0], &idx[3], &idx[1], &idx[4], &idx[2], &idx[5]);
                if (matches != 6) {
                    printf("File can't be read by our simple parser : ( Try exporting with other options\n");
                    return;
                }
                tempTexelIndex.push_back(idx[3] - 1);
                tempTexelIndex.push_back(idx[4] - 1);
                tempTexelIndex.push_back(idx[5] - 1);
            }
            else if (normNum > 0) {
                int matches = fscanf(file, "%d//%d %d//%d %d//%d\n", &idx[0], &idx[6], &idx[1], &idx[7], &idx[2], &idx[8]);
                if (matches != 6) {
                    printf("File can't be read by our simple parser : ( Try exporting with other options\n");
                    return;
                }
                tempNormIndex.push_back(idx[6] - 1);
                tempNormIndex.push_back(idx[7] - 1);
                tempNormIndex.push_back(idx[8] - 1);
            }
            else {
                int matches = fscanf(file, "%d %d %d\n", &idx[0], &idx[1], &idx[2]);
                if (matches != 3) {
                    printf("File can't be read by our simple parser : ( Try exporting with other options\n");
                    return;
                }
            }
            tempPosIndex.push_back(idx[0] - 1);
            tempPosIndex.push_back(idx[1] - 1);
            tempPosIndex.push_back(idx[2] - 1);
            indexNum++;
        }
    }


    if (posNum > 0)  Attribute::init(pos, tempPos.data(), tempPosIndex.data(), posNum, indexNum, 3);
    if (texelNum > 0)  Attribute::init(texel, tempTexel.data(), tempTexelIndex.data(), texelNum, indexNum, 2);
    if (normNum > 0)  Attribute::init(normal, tempNorm.data(), tempNormIndex.data(), normNum, indexNum, 3);
}

__global__ void Shrink(const Attribute attr, float* buf, float s) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    if (px >= attr.vaoNum)return;
    int idx0 = attr.vao[px * 3];
    int idx1 = attr.vao[px * 3 + 1];
    int idx2 = attr.vao[px * 3 + 2];
    for (int i = 0; i < attr.dimention; i++) {
        atomicAdd(&attr.vbo[idx0 * attr.dimention + i], (buf[idx1 * attr.dimention + i]- buf[idx0 * attr.dimention + i]) * s);
        atomicAdd(&attr.vbo[idx1 * attr.dimention + i], (buf[idx2 * attr.dimention + i]- buf[idx1 * attr.dimention + i]) * s);
        atomicAdd(&attr.vbo[idx2 * attr.dimention + i], (buf[idx0 * attr.dimention + i]- buf[idx2 * attr.dimention + i]) * s);
    }
}

void Attribute::posShrink(Attribute& pos, float s, int repeat) {
    dim3 block = dim3(pos.vaoNum > 1024 ? 1024 : pos.vaoNum);
    dim3 grid = dim3((pos.vaoNum - 1) / 1024 + 1);
    float* buf;
    hipMalloc(&buf, pos.vboNum * pos.dimention * sizeof(float));
    void* args[] = { &pos,&buf, &s};
    for (int i = 0; i < repeat; i++) {
        hipMemcpy(buf, pos.vbo, pos.vboNum * pos.dimention * sizeof(float), hipMemcpyDeviceToDevice);
        hipLaunchKernel(reinterpret_cast<const void*>(Shrink), block, grid, args, 0, NULL);
    }
    hipFree(buf);
}